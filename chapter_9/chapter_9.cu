#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUM_BINS 26
#define COARSE_FACTOR 4

__global__ void histo_private_kernel(char *data, unsigned int length, unsigned int *histo)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length)
    {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26)
        {
            atomicAdd(&(histo[blockIdx.x * NUM_BINS + alphabet_position / 4]), 1);
        }
    }
    if (blockIdx.x > 0)
    {
        __syncthreads();
        for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x)
        {
            unsigned int binValue = histo[blockIdx.x * NUM_BINS + bin];
            if (binValue > 0)
            {
                atomicAdd(&(histo[bin]), binValue);
            }
        }
    }
}

__global__ void histo_private_kernel_shared(char *data, unsigned int length, unsigned int *histo)
{
    // initialize privatized bins
    __shared__ unsigned int histo_s[NUM_BINS];
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x)
    {
        histo_s[bin] = 0u;
    }
    __syncthreads();

    // Histogram calculation
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length)
    {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26)
        {
            atomicAdd(&(histo[blockIdx.x * NUM_BINS + alphabet_position / 4]), 1);
        }
    }
    __syncthreads();

    // Reduction, commit to global memory
    if (blockIdx.x > 0)
    {
        __syncthreads();
        for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x)
        {
            unsigned int binValue = histo[blockIdx.x * NUM_BINS + bin];
            if (binValue > 0)
            {
                atomicAdd(&(histo[bin]), binValue);
            }
        }
    }
}

__global__ void histo_private_kernel_shared_coarsened(char *data, unsigned int length, unsigned int *histo)
{
    // initialize privatized bins
    __shared__ unsigned int histo_s[NUM_BINS];
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x)
    {
        histo_s[bin] = 0u;
    }
    __syncthreads();

    // Histogram calculation with coarsening
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned int offset = 0; offset < COARSE_FACTOR && i + offset * blockDim.x * gridDim.x < length; offset++)
    {
        unsigned int idx = i + offset * blockDim.x * gridDim.x;
        int alphabet_position = data[idx] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26)
        {
            atomicAdd(&(histo[blockIdx.x * NUM_BINS + alphabet_position / 4]), 1);
        }
    }
    __syncthreads();

    // Reduction, commit to global memory
    if (blockIdx.x > 0)
    {
        __syncthreads();
        for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x)
        {
            unsigned int binValue = histo[blockIdx.x * NUM_BINS + bin];
            if (binValue > 0)
            {
                atomicAdd(&(histo[bin]), binValue);
            }
        }
    }
}

__global__ void histo_private_kernel_shared_aggregated(char *data, unsigned int length, unsigned int *histo)
{
    // initialize privatized bins
    __shared__ unsigned int histo_s[NUM_BINS];
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x)
    {
        histo_s[bin] = 0u;
    }
    __syncthreads();

    // Histogram calculation
    unsigned int accumulator = 0;
    int prevBinIdx = -1;
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned int i = tid; i < length; i += blockDim.x * gridDim.x)
    {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26)
        {
            int bin = alphabet_position / 4;
            if (bin == prevBinIdx)
            {
                ++accumulator;
            }
            else
            {
                if (accumulator > 0)
                {
                    atomicAdd(&(histo_s[prevBinIdx]), accumulator);
                }
                accumulator = 1;
                prevBinIdx = bin;
            }
        }
    }
    if (accumulator > 0)
    {
        atomicAdd(&(histo_s[prevBinIdx]), accumulator);
    }
    __syncthreads();

    // Reduction, commit to global memory
    if (blockIdx.x > 0)
    {
        __syncthreads();
        for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x)
        {
            unsigned int binValue = histo[blockIdx.x * NUM_BINS + bin];
            if (binValue > 0)
            {
                atomicAdd(&(histo[bin]), binValue);
            }
        }
    }
}

int main()
{
    // Test with different input sizes
    const unsigned int LENGTHS[] = {100, 1000, 10000, 100000, 1000000, 10000000, 100000000, 1000000000};
    const unsigned int LENGTHS_SIZE = sizeof(LENGTHS) / sizeof(LENGTHS[0]);
    for (int test = 0; test < LENGTHS_SIZE; test++)
    {
        const unsigned int LENGTH = LENGTHS[test];
        printf("\nTesting with LENGTH = %u\n", LENGTH);

        // Input data setup
        char *h_data = (char *)malloc(LENGTH * sizeof(char));
        // Fill with random lowercase letters
        for (unsigned int i = 0; i < LENGTH; i++)
        {
            h_data[i] = 'a' + (rand() % 26);
        }

        // Calculate grid dimensions first
        dim3 blockDim(256);
        dim3 gridDim((LENGTH + blockDim.x - 1) / blockDim.x);
        int numBlocks = gridDim.x; // Store grid dimension

        // Allocate device memory
        char *d_data;
        unsigned int *d_histo;
        hipMalloc((void **)&d_data, LENGTH * sizeof(char));
        hipMalloc((void **)&d_histo, NUM_BINS * numBlocks * sizeof(unsigned int));

        // Copy data to device
        hipMemcpy(d_data, h_data, LENGTH * sizeof(char), hipMemcpyHostToDevice);

        // Time first kernel (histo_private_kernel)
        hipEvent_t start1, stop1;
        hipEventCreate(&start1);
        hipEventCreate(&stop1);

        hipMemset(d_histo, 0, NUM_BINS * numBlocks * sizeof(unsigned int));
        hipEventRecord(start1);
        histo_private_kernel<<<gridDim, blockDim>>>(d_data, LENGTH, d_histo);
        hipEventRecord(stop1);

        hipEventSynchronize(stop1);
        float milliseconds1 = 0;
        hipEventElapsedTime(&milliseconds1, start1, stop1);
        printf("histo_private_kernel execution time: %f ms\n", milliseconds1);

        // Time second kernel (histo_private_kernel_shared)
        hipEvent_t start2, stop2;
        hipEventCreate(&start2);
        hipEventCreate(&stop2);

        hipMemset(d_histo, 0, NUM_BINS * numBlocks * sizeof(unsigned int));
        hipEventRecord(start2);
        histo_private_kernel_shared<<<gridDim, blockDim>>>(d_data, LENGTH, d_histo);
        hipEventRecord(stop2);

        hipEventSynchronize(stop2);
        float milliseconds2 = 0;
        hipEventElapsedTime(&milliseconds2, start2, stop2);
        printf("histo_private_kernel_shared execution time: %f ms (%.2fx)\n",
               milliseconds2, milliseconds2 / milliseconds1);

        // Time third kernel (histo_private_kernel_shared_coarsened)
        hipEvent_t start3, stop3;
        hipEventCreate(&start3);
        hipEventCreate(&stop3);

        hipMemset(d_histo, 0, NUM_BINS * numBlocks * sizeof(unsigned int));
        hipEventRecord(start3);
        histo_private_kernel_shared_coarsened<<<gridDim, blockDim>>>(d_data, LENGTH, d_histo);
        hipEventRecord(stop3);

        hipEventSynchronize(stop3);
        float milliseconds3 = 0;
        hipEventElapsedTime(&milliseconds3, start3, stop3);
        printf("histo_private_kernel_shared_coarsened execution time: %f ms (%.2fx)\n",
               milliseconds3, milliseconds3 / milliseconds1);

        // Time fourth kernel (histo_private_kernel_shared_aggregated)
        hipEvent_t start4, stop4;
        hipEventCreate(&start4);
        hipEventCreate(&stop4);

        hipMemset(d_histo, 0, NUM_BINS * numBlocks * sizeof(unsigned int));
        hipEventRecord(start4);
        histo_private_kernel_shared_aggregated<<<gridDim, blockDim>>>(d_data, LENGTH, d_histo);
        hipEventRecord(stop4);

        hipEventSynchronize(stop4);
        float milliseconds4 = 0;
        hipEventElapsedTime(&milliseconds4, start4, stop4);
        printf("histo_private_kernel_shared_aggregated execution time: %f ms (%.2fx)\n",
               milliseconds4, milliseconds4 / milliseconds1);

        // Cleanup
        hipFree(d_data);
        hipFree(d_histo);
        free(h_data);
        hipEventDestroy(start1);
        hipEventDestroy(stop1);
        hipEventDestroy(start2);
        hipEventDestroy(stop2);
        hipEventDestroy(start3);
        hipEventDestroy(stop3);
        hipEventDestroy(start4);
        hipEventDestroy(stop4);
    }

    return 0;
}